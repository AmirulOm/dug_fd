#include <hip/hip_runtime.h>
#include <iostream>
#include <array>
#include <assert.h>

#ifndef SIZE
#define SIZE 10000
#endif

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) 
  if (result != cudaSuccess) {
    std::cerr << "CUDA Runtime Error: " << cudaGetErrorString(result) << "\n";
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

// last block to handle left x-direction stencil computation
__device__ void fd_leftover_kernel(double *a, double *b,const int& row,const int& i,const int& offset,const int& rowOffset)
{
  __shared__ double dfl[3][9];

  if(threadIdx.x < blockIdx.x)
  {
    dfl[0][i] = a[i + (blockIdx.x * blockDim.x) + (rowOffset * row) - offset +  blockDim.x];
    dfl[1][i] = a[i + (blockIdx.x * blockDim.x) + (rowOffset * (row+1)) - offset + blockDim.x];
    dfl[2][i] = a[i + (blockIdx.x * blockDim.x) + (rowOffset * (row+2)) - offset + blockDim.x];
  }

  __syncthreads();

  if(threadIdx.x < blockIdx.x)
  {
    int id = i + (blockIdx.x * blockDim.x) + (rowOffset * (row+1)) - offset + blockDim.x;

    if(i != 0 && i != (9 - 1)) // branch divergence on first and last thread
    { 
      b[id]= dfl[1][i]/2.0 + dfl[1][i+1]/8.0 + dfl[1][i-1]/8.0 +  dfl[0][i]/8.0 +  dfl[2][i]/8.0;
    }
  }
}

__global__ void fd_kernel(double *a, double *b)
{
    __shared__ double df[3][1000];
  
    const int i = threadIdx.x;
    const int offset = blockIdx.x;
    const size_t rowOffset = SIZE; 
    size_t row=0;

    df[0][i] = a[i + (blockIdx.x * blockDim.x) + (rowOffset * row) - offset];
    df[1][i] = a[i + (blockIdx.x * blockDim.x) + (rowOffset * (row+1)) - offset];
   

    for(; row < SIZE-2; ++row)
    {
      df[2][i] = a[i + (blockIdx.x * blockDim.x) + (rowOffset * (row+2)) - offset];
      __syncthreads();

      int id = i + (blockIdx.x * blockDim.x) +  (rowOffset * (row+1) - offset);
      
      if(i != 0 && i != (blockDim.x - 1)) // branch divergence on first and last thread
      {  
        b[id]= df[1][i]/2.0 + df[1][i+1]/8.0 + df[1][i-1]/8.0 +  df[0][i]/8.0 +  df[2][i]/8.0;
      }

      if( blockIdx.x  == (gridDim.x - 1))
        fd_leftover_kernel(a, b, row,i, offset, rowOffset);
      
        
      df[0][i] = df[1][i];
      df[1][i] = df[2][i];
      
    }
}



void fd()
{
    double *a_host = new double[SIZE*SIZE];
    double *a_dev, *b_dev;
    
    memset(a_host,0,sizeof(double)*SIZE*SIZE);

    size_t ind(0);
    const size_t ITER_SIZE = 10;

    for(; ind < SIZE; ++ind) 
    {
            a_host[ind] = 1.0; // Top boundary
            a_host[ind*SIZE] = 3.0; // Top boundary
            a_host[ind*SIZE + SIZE - 1] = 2.0; // Top boundary
            a_host[SIZE * (SIZE-1) + ind] = 4.0; // Top boundary
    }

    // Unnecessary copy fron a to b
    // b=a

    // Allocate device memory
    checkCuda( hipMalloc( (void**)&a_dev, sizeof(double) * SIZE * SIZE ));
    checkCuda( hipMalloc( (void**)&b_dev, sizeof(double) * SIZE * SIZE ));

    // Transfer device memory
    checkCuda( hipMemcpy(a_dev, a_host, sizeof(double) * SIZE * SIZE, hipMemcpyHostToDevice) );  
    checkCuda( hipMemcpy(b_dev, a_host, sizeof(double) * SIZE * SIZE, hipMemcpyHostToDevice) ); 

    dim3 grid(10), block(1000);
    
    for (size_t iter = 0; iter < ITER_SIZE; ++iter)
      if( iter % 2 == 0)
        fd_kernel<<<grid,block>>>(a_dev,b_dev);
      else
        fd_kernel<<<grid,block>>>(b_dev,a_dev);
    checkCuda( hipPeekAtLastError() );

    if((ITER_SIZE-1) % 2 == 0)
      checkCuda( hipMemcpy(a_host, b_dev, sizeof(double) * SIZE * SIZE, hipMemcpyDeviceToHost) );  
    else
      checkCuda( hipMemcpy(a_host, a_dev, sizeof(double) * SIZE * SIZE, hipMemcpyDeviceToHost) ); 

    std::cout  <<a_host[4 + (4*SIZE)] << " " << 
                 a_host[999 + (999*SIZE)] << " " <<
                 a_host[9994 + (9994*SIZE)] << "\n";
}

int main()
{
    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, 0) );
    std::cout << "\nDevice Name: \n" << prop.name;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << "\n";
    std::cout << "maxThreadsDim    : " << prop.maxThreadsDim[0] << "," << prop.maxThreadsDim[1] << "," << prop.maxThreadsDim[2] << "\n";
    std::cout << "maxThreadsPerBlock: " << prop.maxThreadsPerBlock << "\n";

    fd();

    return 0;
}